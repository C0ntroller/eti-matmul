#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define BLOCK_SIZE 4
#define HANDLE_ERROR(err) (handleCudaError(err, __FILE__, __LINE__))

static void handleCudaError(hipError_t err, const char *file, int line){
    if(err!=hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

void matmul_cpu(double *A, double *B, double *C, int n) {
    for (int i = 0; i < n; i++)
        for (int k = 0; k < n; k++)
            for (int j = 0; j < n; j++)
                C[i * n + j] += A[i * n + k] * B[k * n + j];
}

int results_correct(double* A, double* B, int n){
    for(int i = 0; i<n*n; i++){
        if((*(A+i) - *(B+i)) > 0.0001){
            return 0;
        }
    }
    return 1;
}

__global__ void matmul_gpu(double *A, double *B, double *C, int n){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < n; j += blockDim.y * gridDim.y) {
            for (int k = 0; k < n; k++) {
                C[i * n + j] += A[i * n + k] * B[k * n + j];
            }
        }
    }

}

int main(int argc, char **argv) {

    if (argc != 2) {
        printf("Please specify matrix size.\n");
        return 0;
    }

    int n;
    sscanf(argv[1], "%d", &n);

    // Allocate and initialize host memory
    double *C_check = (double *) malloc(n * n * sizeof(double));

    // Allocate cuda-managed memory
    double *A, *B, *C;
    HANDLE_ERROR(hipMallocManaged(&A, n*n*sizeof(double)));
    HANDLE_ERROR(hipMallocManaged(&B, n*n*sizeof(double)));
    HANDLE_ERROR(hipMallocManaged(&C, n*n*sizeof(double)));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            A[i * n + j] = (double) ((rand() % 2000) + 9000) / 10000;
            B[i * n + j] = (double) ((rand() % 2000) + 9000) / 10000;
            C[i * n + j] = (double) 0;
            C_check[i * n + j] = (double) 0;
        }
    }

#if 0
    // Run reference implementation on cpu
    matmul_cpu(A, B, C_check, n);
#endif


    // Get device id
    int deviceId;
    HANDLE_ERROR(hipGetDevice(&deviceId));


    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(ceil(n/BLOCK_SIZE), ceil(n/BLOCK_SIZE));

    // Prefetch data to gpu
    HANDLE_ERROR(hipMemPrefetchAsync(A, n*n*sizeof(double), deviceId));
    HANDLE_ERROR(hipMemPrefetchAsync(B, n*n*sizeof(double), deviceId));
    HANDLE_ERROR(hipMemPrefetchAsync(C, n*n*sizeof(double), deviceId));

    hipEvent_t cstart, cend;
    hipStream_t cstream;
    HANDLE_ERROR(hipEventCreate(&cstart));
    HANDLE_ERROR(hipEventCreate(&cend));
    HANDLE_ERROR(hipStreamCreate(&cstream));
    float runtime = 0;

    for(int i = 0; i<10; i++) {

        HANDLE_ERROR(hipDeviceSynchronize());

        HANDLE_ERROR(hipEventRecord(cstart, cstream));

        // Perform matmul on n elements
        matmul_gpu<<<gridSize, blockSize>>>(A, B, C, n);
        HANDLE_ERROR(hipGetLastError());

        // Prefetch data from gpu
        HANDLE_ERROR(hipMemPrefetchAsync(C, n * n * sizeof(double), hipCpuDeviceId));

        HANDLE_ERROR(hipEventRecord(cend, cstream));
        HANDLE_ERROR(hipEventSynchronize(cend));
        HANDLE_ERROR(hipEventElapsedTime(&runtime, cstart, cend));
        printf("%f,", runtime);

        HANDLE_ERROR(hipMemset(C, 0, n * n * sizeof(double)));
    }

    printf("\n");

#if 0
    if(!results_correct(C, C_check, n)){
        printf("incorrect results\n");

        HANDLE_ERROR(hipFree(A));
        HANDLE_ERROR(hipFree(B));
        HANDLE_ERROR(hipFree(C));

        free(C_check);

        return 1;
    }
#endif

    HANDLE_ERROR(hipFree(A));
    HANDLE_ERROR(hipFree(B));
    HANDLE_ERROR(hipFree(C));

    free(C_check);

    return 0;
}